#ifdef __cplusplus
extern "C" {
#include <stdio.h>
#include "chplgpu.h"

void gpuAllocate(void **data, size_t size) {
  printf("data = %p and size = %d\n", data, (int)size);
  hipError_t err = hipMalloc( data, size);
  if ( hipSuccess != err) {
    fprintf(stderr, "gpuAllocate() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
  //printf("gpuAllocate: Allocated %u bytes at %p\n", (unsigned int)size, *data);
}

void gpuFree(void **data) {
  hipError_t err = hipFree( *data);
  if ( hipSuccess != err) {
    fprintf(stderr, "gpuFree() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyGPUtoHost(void *dst, void *src, size_t size) {
  //printf("** Copying gpu to host : DST = %p SRC = %p SIZE = %u\n", dst, src, (unsigned int)size);
  hipError_t err = hipMemcpy( dst, src, size, hipMemcpyDeviceToHost);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyGPUtoHost() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyHosttoHost(void *dst, void *src, size_t size) {
  //printf("** Copying gpu to host : DST = %p SRC = %p SIZE = %u\n", dst, src, (unsigned int)size);
  hipError_t err = hipMemcpy( dst, src, size, hipMemcpyHostToHost);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyGPUtoHost() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyGPUtoGPU(void *dst, void *src, size_t size) {
  hipError_t err = hipMemcpy( dst, src, size, hipMemcpyDeviceToDevice);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyGPUtoGPU() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}
void copyHostToGPU(void *dst, void *src, size_t size) {
  printf("** Copying host to gpu : DST = %p SRC = %p SIZE = %u\n", dst, src, (unsigned int)size);
  hipError_t err = hipMemcpy( dst, src, size, hipMemcpyHostToDevice);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyHostToGPU() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyToConstMem(char *symbol, void *src, size_t size) {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(symbol), src, size, 0, hipMemcpyHostToDevice);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyToConstMem() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void chpl_init_accelerator(void) {
  hipSetDevice( 0 ); // init device 0 (this is not clean)
}

}
#endif
