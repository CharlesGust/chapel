#include "hip/hip_runtime.h"
#ifdef __cplusplus
#include <typeinfo>
enum ReduceOp {
    MAX,
    MIN,
    SUM,
    PROD,
    BITAND,
    BITOR,
    BITXOR,
    LOGAND,
    LOGOR
  };

#include "sharedmem.cuh"
template <class T>
void 
cuda_reduce(int size, int threads, int blocks, 
    int whichKernel, const ReduceOp op, T *d_idata, T *d_odata);

template <class T, unsigned int blockSize>
__global__ void
reduce5(ReduceOp op, T *g_idata, T *g_odata, unsigned int n);

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(ReduceOp op, T *g_idata, T *g_odata, unsigned int n);

extern "C" {
#include <stdio.h>
#include "chplgpu.h"

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

  bool isPow2(unsigned int x)
  {
    return ((x&(x-1))==0);
  }

  unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
  }

  ////////////////////////////////////////////////////////////////////////////////
  // Compute the number of threads and blocks to use for the given reduction kernel
  // For the kernels >= 3, we set threads / block to the minimum of maxThreads and
  // n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
  // 6, we observe the maximum specified number of blocks, because each thread in 
  // that kernel can process a variable number of elements.
  ////////////////////////////////////////////////////////////////////////////////
#define MAXBLOCKS 64
#define MAXTHREADS 128
  void getNumBlocksAndThreads(int whichKernel, int n, int &blocks, int &threads)
  {
    threads = (n < MAXTHREADS*2) ? nextPow2((n + 1)/ 2) : MAXTHREADS;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);
    if (whichKernel == 6)
      blocks = min(MAXBLOCKS, blocks);
  }
}

  template <class T>
T _internalReduce_real(int  n, 
    int  numThreads,
    int  numBlocks,
    int whichKernel,
    const ReduceOp op,
    T* h_odata,
    T* d_idata, 
    T* d_odata)
{
  T gpu_result = 0;
  bool needReadBack = true;
  gpu_result = 0;

  hipDeviceSynchronize();

  // execute the kernel
  cuda_reduce<T>(n, numThreads, numBlocks, whichKernel, op, d_idata, d_odata);

  // check if kernel execution generated an error
  //cutilCheckMsg("Kernel execution failed");

  // sum partial block sums on GPU
  int s=numBlocks;
  int kernel = (whichKernel == 6) ? 5 : whichKernel;
  while(s > 1) 
  {
    int threads = 0, blocks = 0;
    getNumBlocksAndThreads(kernel, s, blocks, threads);

    cuda_reduce<T>(s, threads, blocks, kernel, op, d_odata, d_odata);

    s = (s + (threads*2-1)) / (threads*2);
  }

  if (s > 1)
  {
    // copy result from device to host
    hipMemcpy( h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost);
    switch (op) {
      case MIN :
        for(int i=0; i < s; i++) gpu_result = min(gpu_result, h_odata[i]);
        break;
      case MAX :
        for(int i=0; i < s; i++) gpu_result = max(gpu_result, h_odata[i]);
        break;
      case SUM :
        for(int i=0; i < s; i++) gpu_result += h_odata[i];
        break;
      case PROD:
        for(int i=0; i < s; i++) gpu_result *= h_odata[i];
        break;
      case LOGAND:
        for(int i=0; i < s; i++) gpu_result = gpu_result && h_odata[i];
        break;
      case LOGOR:
        for(int i=0; i < s; i++) gpu_result = gpu_result || h_odata[i];
        break;
    }

    needReadBack = false;
  }

  hipDeviceSynchronize();

  if (needReadBack)
  {
    // copy final sum from device to host
    hipMemcpy( &gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost);
  }

  return gpu_result;
}


  template <class T>
T _internalReduce_int(int  n, 
    int  numThreads,
    int  numBlocks,
    int whichKernel,
    const ReduceOp op,
    T* h_odata,
    T* d_idata, 
    T* d_odata)
{
  T gpu_result = 0;
  bool needReadBack = true;
  gpu_result = 0;

  hipDeviceSynchronize();

  // execute the kernel
  cuda_reduce<T>(n, numThreads, numBlocks, whichKernel, op, d_idata, d_odata);

  // check if kernel execution generated an error
  //cutilCheckMsg("Kernel execution failed");

  // sum partial block sums on GPU
  int s=numBlocks;
  int kernel = (whichKernel == 6) ? 5 : whichKernel;
  while(s > 1) 
  {
    int threads = 0, blocks = 0;
    getNumBlocksAndThreads(kernel, s, blocks, threads);

    cuda_reduce<T>(s, threads, blocks, kernel, op, d_odata, d_odata);

    s = (s + (threads*2-1)) / (threads*2);
  }

  if (s > 1)
  {
    // copy result from device to host
    hipMemcpy( h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost);
    switch (op) {
      case MIN :
        for(int i=0; i < s; i++) gpu_result = min(gpu_result, h_odata[i]);
        break;
      case MAX :
        for(int i=0; i < s; i++) gpu_result = max(gpu_result, h_odata[i]);
        break;
      case SUM :
        for(int i=0; i < s; i++) gpu_result += h_odata[i];
        break;
      case PROD:
        for(int i=0; i < s; i++) gpu_result *= h_odata[i];
        break;
      case BITAND:
  //      if (!isFloat)
  //        for(int i=0; i < s; i++) gpu_result &= h_odata[i];
        break;
      case BITOR:
  //      if (!isFloat)
  //        for(int i=0; i < s; i++) gpu_result |= h_odata[i];
        break;
      case BITXOR:
  //      if (!isFloat)
  //        for(int i=0; i < s; i++) gpu_result ^= h_odata[i];
        break;
      case LOGAND:
        for(int i=0; i < s; i++) gpu_result = gpu_result && h_odata[i];
        break;
      case LOGOR:
        for(int i=0; i < s; i++) gpu_result = gpu_result || h_odata[i];
        break;
    }

    needReadBack = false;
  }

  hipDeviceSynchronize();

  if (needReadBack)
  {
    // copy final sum from device to host
    hipMemcpy( &gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost);
  }

  return gpu_result;
}

template <class T>
  void 
cuda_reduce(int size, int threads, int blocks, 
    int whichKernel, const ReduceOp op, T *d_idata, T *d_odata)
{
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);
  int smemSize = threads * sizeof(T);

  // choose which of the optimized versions of reduction to launch
  switch (whichKernel)
  {
    case 5:
      switch (threads)
      {
        case 512:
          reduce5<T, 512><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case 256:
          reduce5<T, 256><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case 128:
          reduce5<T, 128><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case 64:
          reduce5<T, 64><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case 32:
          reduce5<T, 32><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case 16:
          reduce5<T, 16><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case  8:
          reduce5<T, 8><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case  4:
          reduce5<T, 4><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case  2:
          reduce5<T, 2><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        case  1:
          reduce5<T, 1><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
      }
      break;       
    case 6:
    default:
      if (isPow2(size))
      {
        switch (threads)
        {
          case 512:
            reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 256:
            reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 128:
            reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 64:
            reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 32:
            reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 16:
            reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  8:
            reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  4:
            reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  2:
            reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  1:
            reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        }
      }
      else
      {
        switch (threads)
        {
          case 512:
            reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 256:
            reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 128:
            reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 64:
            reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 32:
            reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case 16:
            reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  8:
            reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  4:
            reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  2:
            reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
          case  1:
            reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(op, d_idata, d_odata, size); break;
        }
      }
      break;       
  }
}

  float gpu_special_reduce_real32(float *src, int size, const char *op) {
    int numBlocks = 0;
    int numThreads = 0;
    int whichKernel = 6;
    ReduceOp opType;

    if (!strcasecmp(op, "Max"))
      opType = MAX;
    else if (!strcasecmp(op, "Min"))
      opType = MIN;
    else if (!strcasecmp(op, "Sum"))
      opType = SUM;
    else if (!strcasecmp(op, "Product"))
      opType = PROD;
    else if (!strcasecmp(op, "LogicalAnd"))
      opType = LOGAND;
    else if (!strcasecmp(op, "LogicalOr"))
      opType = LOGOR;
    else if (!strcasecmp(op, "BitwiseAnd"))
      opType = BITAND;
    else if (!strcasecmp(op, "BitwiseOr"))
      opType = BITOR;
    else if (!strcasecmp(op, "BitwiseXor"))
      opType = BITXOR;

    getNumBlocksAndThreads(whichKernel, size, numBlocks, numThreads);

  // allocate mem for the result on host side
  float* h_odata = (float*) malloc(numBlocks*sizeof(float));
  // allocate device memory and data
  float* d_odata = NULL;

  hipMalloc((void**) &d_odata, numBlocks*sizeof(float));

  // copy data directly to device memory
  hipMemcpy(d_odata, src, numBlocks*sizeof(float), hipMemcpyDeviceToDevice);

  float gpu_result = _internalReduce_real<float>(size, numThreads, numBlocks, whichKernel, opType,
      h_odata, src, d_odata);

  return gpu_result;

  }

  double gpu_special_reduce_real64(double *src, int size, const char *op) {
    int numBlocks = 0;
    int numThreads = 0;
    int whichKernel = 6;

    ReduceOp opType = MAX;
    printf("REDUCE : op type = %s\n", op);
    if (!strcasecmp(op, "Min"))
      opType = MIN;
    else if (!strcasecmp(op, "Sum"))
      opType = SUM;
    else if (!strcasecmp(op, "Product"))
      opType = PROD;
    else if (!strcasecmp(op, "LogicalAnd"))
      opType = LOGAND;
    else if (!strcasecmp(op, "LogicalOr"))
      opType = LOGOR;
    else if (!strcasecmp(op, "BitwiseAnd"))
      opType = BITAND;
    else if (!strcasecmp(op, "BitwiseOr"))
      opType = BITOR;
    else if (!strcasecmp(op, "BitwiseXor"))
      opType = BITXOR;

    getNumBlocksAndThreads(whichKernel, size, numBlocks, numThreads);

  // allocate mem for the result on host side
  double* h_odata = (double*) malloc(numBlocks*sizeof(double));
  // allocate device memory and data
  double* d_odata = NULL;

  hipMalloc((void**) &d_odata, numBlocks*sizeof(double));

  // copy data directly to device memory
  hipMemcpy(d_odata, src, numBlocks*sizeof(double), hipMemcpyDeviceToDevice);

  double gpu_result = _internalReduce_real<double>(size, numThreads, numBlocks, whichKernel, opType,
      h_odata, src, d_odata);

  return gpu_result;

}

int32_t gpu_special_reduceint32_t(int32_t *src, int size, const char *op) {
  int numBlocks = 0;
  int numThreads = 0;
  int whichKernel = 6;
  ReduceOp opType;

  if (!strcasecmp(op, "Max"))
    opType = MAX;
  else if (!strcasecmp(op, "Min"))
    opType = MIN;
  else if (!strcasecmp(op, "Sum"))
    opType = SUM;
  else if (!strcasecmp(op, "Product"))
    opType = PROD;
  else if (!strcasecmp(op, "LogicalAnd"))
    opType = LOGAND;
  else if (!strcasecmp(op, "LogicalOr"))
    opType = LOGOR;
  else if (!strcasecmp(op, "BitwiseAnd"))
    opType = BITAND;
  else if (!strcasecmp(op, "BitwiseOr"))
    opType = BITOR;
  else if (!strcasecmp(op, "BitwiseXor"))
    opType = BITXOR;

  getNumBlocksAndThreads(whichKernel, size, numBlocks, numThreads);

  // allocate mem for the result on host side
  int32_t* h_odata = (int32_t*) malloc(numBlocks*sizeof(int32_t));
  // allocate device memory and data
  int32_t* d_odata = NULL;

  hipMalloc((void**) &d_odata, numBlocks*sizeof(int32_t));

  // copy data directly to device memory
  hipMemcpy(d_odata, src, numBlocks*sizeof(int32_t), hipMemcpyDeviceToDevice);

  int32_t gpu_result = _internalReduce_int<int32_t>(size, numThreads, numBlocks, whichKernel, opType,
      h_odata, src, d_odata);

  return gpu_result;
}

template <class T, unsigned int blockSize>
  __global__ void
reduce5(ReduceOp op, T *g_idata, T *g_odata, unsigned int n)
{
  SharedMemory<T> smem;
  T *sdata = smem.getPointer();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;
  switch (op) {
    case SUM :
      if (i + blockSize < n) 
        sdata[tid] += g_idata[i+blockSize];  

      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
      if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
      }
      break;
    case PROD :
      if (i + blockSize < n) 
        sdata[tid] *= g_idata[i+blockSize];  

      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] *= sdata[tid + 256]; } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] *= sdata[tid + 128]; } __syncthreads(); }
      if (blockSize >= 128) { if (tid <  64) { sdata[tid] *= sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] *= sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] *= sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] *= sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] *= sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] *= sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] *= sdata[tid +  1]; EMUSYNC; }
      }
      break;
    case MAX :
      if (i + blockSize < n) 
        sdata[tid] = max(sdata[tid], g_idata[i+blockSize]);  

      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
      if (blockSize >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] = max(sdata[tid], sdata[tid + 32]); EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = max(sdata[tid], sdata[tid + 16]); EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = max(sdata[tid], sdata[tid + 8]); EMUSYNC; }
        if (blockSize >=  8) { sdata[tid] = max(sdata[tid], sdata[tid + 4]); EMUSYNC; }
        if (blockSize >=  4) { sdata[tid] = max(sdata[tid], sdata[tid + 2]); EMUSYNC; }
        if (blockSize >=  2) { sdata[tid] = max(sdata[tid], sdata[tid + 1]); EMUSYNC; }
      }
      break;
    case MIN:
      if (i + blockSize < n) 
        sdata[tid] = min(sdata[tid], g_idata[i+blockSize]);  

      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
      if (blockSize >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] = min(sdata[tid], sdata[tid + 32]); EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = min(sdata[tid], sdata[tid + 16]); EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = min(sdata[tid], sdata[tid + 8]); EMUSYNC; }
        if (blockSize >=  8) { sdata[tid] = min(sdata[tid], sdata[tid + 4]); EMUSYNC; }
        if (blockSize >=  4) { sdata[tid] = min(sdata[tid], sdata[tid + 2]); EMUSYNC; }
        if (blockSize >=  2) { sdata[tid] = min(sdata[tid], sdata[tid + 1]); EMUSYNC; }
      }
      break;
  }

  // write result for this block to global mem 
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
   This version adds multiple elements per thread sequentially.  This reduces the overall
   cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
   (Brent's Theorem optimization)
 */
template <class T, unsigned int blockSize, bool nIsPow2>
  __global__ void
reduce6(ReduceOp op, T *g_idata, T *g_odata, unsigned int n)
{
  SharedMemory<T> smem;
  T *sdata = smem.getPointer();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  // we reduce multiple elements per thread.  The number is determined by the 
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  switch (op)
  {
    case SUM :
      sdata[tid] = 0;
      while (i < n)
      {         
        sdata[tid] += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
          sdata[tid] += g_idata[i+blockSize];  
        i += gridSize;
      } 
      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
      if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
      }
      break;
    case MAX :
      sdata[tid] = INT_MIN;
      while (i < n)
      {         
        sdata[tid] = max(sdata[tid], g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
          sdata[tid] = max(sdata[tid], g_idata[i+blockSize]);
        i += gridSize;
      } 
      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] = max(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] = max(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
      if (blockSize >= 128) { if (tid < 64) { sdata[tid] = max(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] = max(sdata[tid], sdata[tid + 32]); EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = max(sdata[tid], sdata[tid + 16]); EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = max(sdata[tid], sdata[tid + 8]); EMUSYNC; }
        if (blockSize >=  8) { sdata[tid] = max(sdata[tid], sdata[tid + 4]); EMUSYNC; }
        if (blockSize >=  4) { sdata[tid] = max(sdata[tid], sdata[tid + 2]); EMUSYNC; }
        if (blockSize >=  2) { sdata[tid] = max(sdata[tid], sdata[tid + 1]); EMUSYNC; }
      }
      break;
    case MIN:
      sdata[tid] = INT_MAX;
      while (i < n)
      {         
        sdata[tid] = min(sdata[tid], g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
          sdata[tid] = min(sdata[tid], g_idata[i+blockSize]);
        i += gridSize;
      } 
      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) { if (tid < 256) { sdata[tid] = min(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
      if (blockSize >= 256) { if (tid < 128) { sdata[tid] = min(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
      if (blockSize >= 128) { if (tid < 64) { sdata[tid] = min(sdata[tid], sdata[tid + 64]); } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
      if (tid < 32)
#endif
      {
        if (blockSize >=  64) { sdata[tid] = min(sdata[tid], sdata[tid + 32]); EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = min(sdata[tid], sdata[tid + 16]); EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = min(sdata[tid], sdata[tid + 8]); EMUSYNC; }
        if (blockSize >=  8) { sdata[tid] = min(sdata[tid], sdata[tid + 4]); EMUSYNC; }
        if (blockSize >=  4) { sdata[tid] = min(sdata[tid], sdata[tid + 2]); EMUSYNC; }
        if (blockSize >=  2) { sdata[tid] = min(sdata[tid], sdata[tid + 1]); EMUSYNC; }
      }
      break;
  }

  // write result for this block to global mem 
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

#endif
