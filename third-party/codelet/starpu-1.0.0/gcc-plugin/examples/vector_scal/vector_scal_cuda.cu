#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2012 Institut National de Recherche en Informatique et Automatique
 * Copyright (C) 2010, 2011  Centre National de la Recherche Scientifique
 * Copyright (C) 2010  Université de Bordeaux 1
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* CUDA implementation of the `vector_scal' task.  */

#include <starpu.h>
#include <starpu_cuda.h>
#include <stdlib.h>

static __global__ void
vector_mult_cuda (float *val, unsigned n, float factor)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    val[i] *= factor;
}

extern "C" void
vector_scal_cuda (size_t size, float vector[], float factor)
{
  unsigned threads_per_block = 64;
  unsigned nblocks = (size + threads_per_block - 1) / threads_per_block;

  vector_mult_cuda <<< nblocks, threads_per_block, 0,
    starpu_cuda_get_local_stream () >>> (vector, size, factor);

  hipStreamSynchronize (starpu_cuda_get_local_stream ());
}
