#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010, 2011  Centre National de la Recherche Scientifique
 * Copyright (C) 2010, 2011  Université de Bordeaux 1
 *
 * Redistribution  and  use  in  source and binary forms, with or without
 * modification,  are  permitted  provided  that the following conditions
 * are met:
 *
 * * Redistributions  of  source  code  must  retain  the above copyright
 *   notice,  this  list  of  conditions  and  the  following  disclaimer.
 * * Redistributions  in  binary  form must reproduce the above copyright
 *   notice,  this list of conditions and the following disclaimer in the
 *   documentation  and/or other materials provided with the distribution.
 * * The name of the author may not be used to endorse or promote products
 *   derived from this software without specific prior written permission.
 *
 * THIS  SOFTWARE  IS  PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * ``AS IS''  AND  ANY  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED  TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A  PARTICULAR  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL
 * SPECIAL,  EXEMPLARY,  OR  CONSEQUENTIAL  DAMAGES  (INCLUDING,  BUT NOT
 * LIMITED  TO,  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE
 * DATA,  OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY  OF  LIABILITY,  WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF  THIS  SOFTWARE,  EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <starpu.h>
#include <starpu_cuda.h>

static __global__ void vector_mult_cuda(float *val, unsigned n, float factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;
        if (i < n)
               val[i] *= factor;
}

extern "C" void scal_cuda_func(void *buffers[], void *_args)
{
        float *factor = (float *)_args;

        /* length of the vector */
        unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
        /* local copy of the vector pointer */
        float *val = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
        unsigned threads_per_block = 64;
        unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        vector_mult_cuda<<<nblocks,threads_per_block, 0, starpu_cuda_get_local_stream()>>>(val, n, *factor);

        hipStreamSynchronize(starpu_cuda_get_local_stream());
}

